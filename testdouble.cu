#include "hip/hip_runtime.h"
#include "mainwindow.h"
#include "qcustomplot.h"
#include "ui_mainwindow.h"

__global__ void TestDouble(double *arr, float *device_double_time, int size) {

    int xid = blockIdx.x * blockDim.x + threadIdx.x;

    clock_t start = clock();

    if (xid < size) {
        for (int i = 0; i < 256; i++)
            arr[xid] = arr[xid] * arr[xid] + arr[xid];
    }

    clock_t end = clock();
    device_double_time[xid] = (float)(end - start);

}
