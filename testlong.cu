#include "hip/hip_runtime.h"
#include "mainwindow.h"
#include "qcustomplot.h"
#include "ui_mainwindow.h"

__global__ void TestLong(long long *arr, float *device_long_time, int size) {

    int xid = blockIdx.x * blockDim.x + threadIdx.x;

    clock_t start = clock();

    if (xid < size) {
        for (int i = 0; i < 256; i++)
            arr[xid] = arr[xid] * arr[xid] + arr[xid];
    }

    clock_t end = clock();
    device_long_time[xid] = (float)(end - start);

}
