#include "hip/hip_runtime.h"
#include "mainwindow.h"
#include "qcustomplot.h"
#include "ui_mainwindow.h"

__global__ void TestInt(int *arr, float *device_int_time, int size) {

    int xid = blockIdx.x * blockDim.x + threadIdx.x;

    clock_t start = clock();

    if (xid < size) {
        for (int i = 0; i < 256; i++)
            arr[xid] = arr[xid] * arr[xid] + arr[xid];
    }

    clock_t end = clock();
    device_int_time[xid] = (float)(end - start);

}
