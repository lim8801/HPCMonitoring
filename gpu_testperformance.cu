#include "mainwindow.h"
#include "qcustomplot.h"
#include "ui_mainwindow.h"

int MainWindow::GPU_TestPerformance(int device, int type) {

    int size = gpu_info[device].SMCount * gpu_info[device].MaxThreadsBlock;

    hipSetDevice(device);

    if (type == 0) {

        host_arr_double = new double[size];

        for (int i = 0; i < size; i++) {
            srand(time(NULL));
            host_arr_double[i] = (double)(rand() % 100) / (double)1000;
        }

        double *device_arr_double;
        hipMalloc((void **)&device_arr_double, sizeof(double) * size);
        hipMemcpy(device_arr_double, host_arr_double, sizeof(double) * size, hipMemcpyHostToDevice);

        float *device_double_time;
        hipMalloc((void **)&device_double_time, sizeof(float) * size);

        TestDouble <<< gpu_info[device].SMCount, gpu_info[device].MaxThreadsBlock >>> (device_arr_double, device_double_time, size);

        hipMemcpy(gpu_info[device].double_time, device_double_time, sizeof(float) * size, hipMemcpyDeviceToHost);

        for (int i = 0; i < gpu_info[device].SMCount; i++) {
            gpu_info[device].double_time_total[i] = 0.0;
            for (int j = 0; j < gpu_info[device].MaxThreadsBlock; j++) {
                gpu_info[device].double_time_total[i] += (gpu_info[device].double_time[i * gpu_info[device].MaxThreadsBlock + j]) / gpu_info[device].ClockRate;
            }
        }

        delete[] host_arr_double;
        hipFree(device_arr_double);
        hipFree(device_double_time);

    } else if (type == 1) {

        host_arr_float = new float[size];

        for (int i = 0; i < size; i++) {
            srand(time(NULL));
            host_arr_float[i] = (float)(rand() % 100) / (float)1000;
        }

        float *device_arr_float;
        hipMalloc((void **)&device_arr_float, sizeof(float) * size);
        hipMemcpy(device_arr_float, host_arr_float, sizeof(float) * size, hipMemcpyHostToDevice);

        float *device_float_time;
        hipMalloc((void **)&device_float_time, sizeof(float) * size);

        TestSingle <<< gpu_info[device].SMCount, gpu_info[device].MaxThreadsBlock >>> (device_arr_float, device_float_time, size);

        hipMemcpy(gpu_info[device].float_time, device_float_time, sizeof(float) * size, hipMemcpyDeviceToHost);

        for (int i = 0; i < gpu_info[device].SMCount; i++) {
            gpu_info[device].float_time_total[i] = 0.0;
            for (int j = 0; j < gpu_info[device].MaxThreadsBlock; j++) {
                gpu_info[device].float_time_total[i] += (gpu_info[device].float_time[i * gpu_info[device].MaxThreadsBlock + j]) / gpu_info[device].ClockRate;
            }
        }

        delete[] host_arr_float;
        hipFree(device_arr_float);
        hipFree(device_float_time);

    } else if (type == 2) {

        host_arr_int = new int[size];

        for (int i = 0; i < size; i++) {
            srand(time(NULL));
            host_arr_int[i] = rand() % 100;
        }

        int *device_arr_int;
        hipMalloc((void **)&device_arr_int, sizeof(int) * size);
        hipMemcpy(device_arr_int, host_arr_int, sizeof(int) * size, hipMemcpyHostToDevice);

        float *device_int_time;
        hipMalloc((void **)&device_int_time, sizeof(float) * size);

        TestInt <<< gpu_info[device].SMCount, gpu_info[device].MaxThreadsBlock >>> (device_arr_int, device_int_time, size);

        hipMemcpy(gpu_info[device].int_time, device_int_time, sizeof(float) * size, hipMemcpyDeviceToHost);

        for (int i = 0; i < gpu_info[device].SMCount; i++) {
            gpu_info[device].int_time_total[i] = 0.0;
            for (int j = 0; j < gpu_info[device].MaxThreadsBlock; j++) {
                gpu_info[device].int_time_total[i] += (gpu_info[device].int_time[i * gpu_info[device].MaxThreadsBlock + j]) / gpu_info[device].ClockRate;
            }
        }

        delete[] host_arr_int;
        hipFree(device_arr_int);
        hipFree(device_int_time);

    } else if (type == 3) {

        host_arr_long = new long long[size];

        for (int i = 0; i < size; i++) {
            srand(time(NULL));
            host_arr_long[i] = rand() % 100;
        }

        long long *device_arr_long;
        hipMalloc((void **)&device_arr_long, sizeof(long long) * size);
        hipMemcpy(device_arr_long, host_arr_long, sizeof(long long) * size, hipMemcpyHostToDevice);

        float *device_long_time;
        hipMalloc((void **)&device_long_time, sizeof(float) * size);

        TestLong <<< gpu_info[device].SMCount, gpu_info[device].MaxThreadsBlock >>> (device_arr_long, device_long_time, size);

        hipMemcpy(gpu_info[device].long_time, device_long_time, sizeof(float) * size, hipMemcpyDeviceToHost);

        for (int i = 0; i < gpu_info[device].SMCount; i++) {
            gpu_info[device].long_time_total[i] = 0.0;
            for (int j = 0; j < gpu_info[device].MaxThreadsBlock; j++) {
                gpu_info[device].long_time_total[i] += (gpu_info[device].long_time[i * gpu_info[device].MaxThreadsBlock + j]) / gpu_info[device].ClockRate;
            }
        }

        delete[] host_arr_long;
        hipFree(device_arr_long);
        hipFree(device_long_time);

    }

    return 0;

}
