#include "hip/hip_runtime.h"
#include "mainwindow.h"
#include "qcustomplot.h"
#include "ui_mainwindow.h"

__global__ void TestDeviceMemcpy(double *d_arr_double, float *d_arr_float, int *d_arr_int, long long *d_arr_long,
                                 float *time_g2s, float *time_s2g, float *time_s2r, float *time_r2s, float *time_g2r, float *time_r2g,
                                 int size) {

    int xid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ double s_arr_double[1024];
    __shared__ float s_arr_float[1024];
    __shared__ int s_arr_int[1024];
    __shared__ long long s_arr_long[1024];

    double r_arr_double;
    float r_arr_float;
    int r_arr_int;
    long long r_arr_long;

    if (xid < size) {

        // g2s
        clock_t start = clock();
        for (int i = 0; i < 256; i++) {
            s_arr_double[threadIdx.x] = d_arr_double[xid];
            s_arr_float[threadIdx.x] = d_arr_float[xid];
            s_arr_int[threadIdx.x] = d_arr_int[xid];
            s_arr_long[threadIdx.x] = d_arr_long[xid];
        }
        clock_t end = clock();
        time_g2s[xid] = (float)(end - start);

        // s2g
        start = clock();
        for (int i = 0; i < 256; i++) {
            d_arr_double[xid] = s_arr_double[threadIdx.x];
            d_arr_float[xid] = s_arr_float[threadIdx.x];
            d_arr_int[xid] = s_arr_int[threadIdx.x];
            d_arr_long[xid] = s_arr_long[threadIdx.x];
        }
        end = clock();
        time_s2g[xid] = (float)(end - start);

        // s2r
        start = clock();
        for (int i = 0; i < 256; i++) {
            r_arr_double = s_arr_double[threadIdx.x];
            r_arr_float = s_arr_float[threadIdx.x];
            r_arr_int = s_arr_int[threadIdx.x];
            r_arr_long = s_arr_long[threadIdx.x];
        }
        end = clock();
        time_s2r[xid] = (float)(end - start);

        // r2s
        start = clock();
        for (int i = 0; i < 256; i++) {
            s_arr_double[threadIdx.x] = r_arr_double;
            s_arr_float[threadIdx.x] = r_arr_float;
            s_arr_int[threadIdx.x] = r_arr_int;
            s_arr_long[threadIdx.x] = r_arr_long;
        }
        end = clock();
        time_r2s[xid] = (float)(end - start);

        // g2r
        start = clock();
        for (int i = 0; i < 256; i++) {
            r_arr_double = d_arr_double[xid];
            r_arr_float = d_arr_float[xid];
            r_arr_int = d_arr_int[xid];
            r_arr_long = d_arr_long[xid];
        }
        end = clock();
        time_g2r[xid] = (float)(end - start);

        // r2g
        start = clock();
        for (int i = 0; i < 256; i++) {
            d_arr_double[xid] = r_arr_double;
            d_arr_float[xid] = r_arr_float;
            d_arr_int[xid] = r_arr_int;
            d_arr_long[xid] = r_arr_long;
        }
        end = clock();
        time_r2g[xid] = (float)(end - start);

    }

}
