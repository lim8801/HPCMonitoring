#include "mainwindow.h"
#include "qcustomplot.h"
#include "ui_mainwindow.h"

int MainWindow::GPU_TestMemcpy(int device) {

    int size = gpu_info[device].SMCount * gpu_info[device].MaxThreadsBlock;

    hipSetDevice(device);

    hipEvent_t event_start;
    hipEvent_t event_end;

    host_arr_double = new double[size];
    host_arr_float = new float[size];
    host_arr_int = new int[size];
    host_arr_long = new long long[size];

    //hipEvent_t event_start;
    //hipEvent_t event_end;

    for (int i = 0; i < size; i++) {

        srand(time(NULL));
        host_arr_double[i] = (double)(rand() % 100) / (double)1000;

        srand(time(NULL));
        host_arr_float[i] = (float)(rand() % 100) / (float)1000;

        srand(time(NULL));
        host_arr_int[i] = rand() % 100;

        srand(time(NULL));
        host_arr_long[i] = rand() % 100;

    }

    double *device_arr_double;
    float *device_arr_float;
    int *device_arr_int;
    long long *device_arr_long;

    // Memcpy HostToDevice
    hipMalloc((void **)&device_arr_double, sizeof(double) * size);
    hipMalloc((void **)&device_arr_float, sizeof(float) * size);
    hipMalloc((void **)&device_arr_int, sizeof(int) * size);
    hipMalloc((void **)&device_arr_long, sizeof(long long) * size);

    hipEventCreate(&event_start);
    hipEventCreate(&event_end);
    hipEventRecord(event_start, 0);

    for (int i = 0; i < 16; i++) {
        hipMemcpy(device_arr_double, host_arr_double, sizeof(double) * size, hipMemcpyHostToDevice);
        hipMemcpy(device_arr_float, host_arr_float, sizeof(float) * size, hipMemcpyHostToDevice);
        hipMemcpy(device_arr_int, host_arr_int, sizeof(int) * size, hipMemcpyHostToDevice);
        hipMemcpy(device_arr_long, host_arr_long, sizeof(long long) * size, hipMemcpyHostToDevice);
    }

    hipEventRecord(event_end, 0);
    hipEventSynchronize(event_end);
    hipEventElapsedTime(&gpu_info[device].h2d_copy_time, event_start, event_end);
    hipEventDestroy(event_start);
    hipEventDestroy(event_end);


    // Memcpy DeviceToHost
    hipEventCreate(&event_start);
    hipEventCreate(&event_end);
    hipEventRecord(event_start, 0);

    for (int i = 0; i < 16; i++) {
        hipMemcpy(host_arr_double, device_arr_double, sizeof(double) * size, hipMemcpyDeviceToHost);
        hipMemcpy(host_arr_float, device_arr_float, sizeof(float) * size, hipMemcpyDeviceToHost);
        hipMemcpy(host_arr_int, device_arr_int, sizeof(int) * size, hipMemcpyDeviceToHost);
        hipMemcpy(host_arr_long, device_arr_long, sizeof(long long) * size, hipMemcpyDeviceToHost);
    }

    hipEventRecord(event_end, 0);
    hipEventSynchronize(event_end);
    hipEventElapsedTime(&gpu_info[device].d2h_copy_time, event_start, event_end);
    hipEventDestroy(event_start);
    hipEventDestroy(event_end);

    //hipEventCreate(&event_start);
    //hipEventCreate(&event_end);
    //hipEventRecord(event_start, 0);

    float *host_g2s = new float[size];
    float *host_s2g = new float[size];
    float *host_s2r = new float[size];
    float *host_r2s = new float[size];
    float *host_g2r = new float[size];
    float *host_r2g = new float[size];
    for (int i = 0; i < size; i++) {
        host_g2s[i] = 0.0;
        host_s2g[i] = 0.0;
        host_s2r[i] = 0.0;
        host_r2s[i] = 0.0;
        host_g2r[i] = 0.0;
        host_r2g[i] = 0.0;
    }

    float *device_g2s;
    float *device_s2g;
    float *device_s2r;
    float *device_r2s;
    float *device_g2r;
    float *device_r2g;
    hipMalloc((void **)&device_g2s, sizeof(float) * size);
    hipMalloc((void **)&device_s2g, sizeof(float) * size);
    hipMalloc((void **)&device_s2r, sizeof(float) * size);
    hipMalloc((void **)&device_r2s, sizeof(float) * size);
    hipMalloc((void **)&device_g2r, sizeof(float) * size);
    hipMalloc((void **)&device_r2g, sizeof(float) * size);
    hipMemcpy(device_g2s, host_g2s, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(device_s2g, host_s2g, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(device_s2r, host_s2r, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(device_r2s, host_r2s, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(device_g2r, host_g2r, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(device_r2g, host_r2g, sizeof(float) * size, hipMemcpyHostToDevice);

    TestDeviceMemcpy <<< gpu_info[device].SMCount, gpu_info[device].MaxThreadsBlock >>> (device_arr_double, device_arr_float, device_arr_int, device_arr_long,
                                                                                         device_g2s, device_s2g, device_s2r, device_r2s, device_g2r, device_r2g, size);

    hipMemcpy(host_g2s, device_g2s, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(host_s2g, device_s2g, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(host_s2r, device_s2r, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(host_r2s, device_r2s, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(host_g2r, device_g2r, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(host_r2g, device_r2g, sizeof(float) * size, hipMemcpyDeviceToHost);
    gpu_info[device].g2s_copy_time = 0.0;
    gpu_info[device].s2g_copy_time = 0.0;
    gpu_info[device].s2r_copy_time = 0.0;
    gpu_info[device].r2s_copy_time = 0.0;
    gpu_info[device].g2r_copy_time = 0.0;
    gpu_info[device].r2g_copy_time = 0.0;
    for (int i = 0; i < size; i++) {
        gpu_info[device].g2s_copy_time += host_g2s[i] / gpu_info[device].ClockRate;
        gpu_info[device].s2g_copy_time += host_s2g[i] / gpu_info[device].ClockRate;
        gpu_info[device].s2r_copy_time += host_s2r[i] / gpu_info[device].ClockRate;
        gpu_info[device].r2s_copy_time += host_r2s[i] / gpu_info[device].ClockRate;
        gpu_info[device].g2r_copy_time += host_g2r[i] / gpu_info[device].ClockRate;
        gpu_info[device].r2g_copy_time += host_r2g[i] / gpu_info[device].ClockRate;
    }
    //hipEventRecord(event_end, 0);
    //hipEventSynchronize(event_end);
    //hipEventElapsedTime(&gpu_info[device].d2h_copy_time, event_start, event_end);
    //hipEventDestroy(event_start);
    //hipEventDestroy(event_end);

    delete[] host_arr_double;
    delete[] host_arr_float;
    delete[] host_arr_int;
    delete[] host_arr_long;
    delete[] host_g2s;
    delete[] host_s2g;
    delete[] host_s2r;
    delete[] host_r2s;
    delete[] host_g2r;
    delete[] host_r2g;
    hipFree(device_arr_double);
    hipFree(device_arr_float);
    hipFree(device_arr_int);
    hipFree(device_arr_long);
    hipFree(device_g2s);
    hipFree(device_s2g);
    hipFree(device_s2r);
    hipFree(device_r2s);
    hipFree(device_g2r);
    hipFree(device_r2g);

    return 0;

}
